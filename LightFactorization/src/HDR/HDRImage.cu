#include "hip/hip_runtime.h"
#include "HDR/GPUHDRImage.h"

__device__ float sincKernel(float x) {               
  if (fabs(x) < 1.0e-4) return 1.0 ;
  else return(sin(x)/x) ;
}

__global__ void computeCoordinates(float* cartesianCoord, float* sphericalCoord, int width, int height) 
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > width * height) return;

	float u, v, r;
	float midWidth = width/2;
	float midHeight = height/2;
	int pixel = idx;
	int x = pixel % width;
	int y = pixel / width;

	u = (x - midWidth)/midWidth;
	v = (y - midHeight)/midHeight;
	r = sqrtf(u * u + v * v);

	if(r > 1.0) {
			
		cartesianCoord[pixel * 3 + 0] = 0;
		cartesianCoord[pixel * 3 + 1] = 0;
		cartesianCoord[pixel * 3 + 2] = 0;
		sphericalCoord[pixel * 2 + 0] = 0;
		sphericalCoord[pixel * 2 + 1] = 0;
			
	} else {
			
		float phi = atan2(v, u);
		float theta = PI * r;

		if(theta != theta) theta = 0;
		if(phi != phi) phi = 0;

		sphericalCoord[pixel * 2 + 0] = theta;
		sphericalCoord[pixel * 2 + 1] = phi;
		cartesianCoord[pixel * 3 + 0] = sin(theta) * cos(phi);
		cartesianCoord[pixel * 3 + 1] = sin(theta) * sin(phi);
		cartesianCoord[pixel * 3 + 2] = cos(theta);
				
	}

}

void GPUComputeCoordinates2(float* cartesianCoord, float* sphericalCoord, int width, int height) 
{

	int blockSize, numBlocks, imageSize;
	imageSize = width * height;
	if(imageSize > 512)
		blockSize = 512;
	else
		blockSize = imageSize % 512;
	numBlocks = (imageSize / 512) + 1;

	computeCoordinates<<<numBlocks, blockSize>>>(cartesianCoord, sphericalCoord, width, height);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
		printf("HDRImage.cu - GPUComputeCoordinates2: %s\n", hipGetErrorString(error));
	
	hipDeviceSynchronize();
}

__global__ void computeDomegaProduct(float *cartesianCoord, float* sphericalCoord, float *domegaProduct, int width, int height) 
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > width * height) return;
	
	float theta, dx, dy, dz, c, domega;
	int pixel = idx;
	
	theta = sphericalCoord[pixel * 2 + 0]; 
	dx = cartesianCoord[pixel * 3 + 0];
	dy = cartesianCoord[pixel * 3 + 1];
	dz = cartesianCoord[pixel * 3 + 2];
	domega = (2*PI/(float)width)*(2*PI/(float)width)*sincKernel(theta);
	c = 0.282095;
	domegaProduct[pixel * 9 + 0] = c * domega;
	c = 0.488603;
	domegaProduct[pixel * 9 + 1] = c * dy * domega;
	domegaProduct[pixel * 9 + 2] = c * dz * domega;
	domegaProduct[pixel * 9 + 3] = c * dx * domega;
	c = 1.092548;
	domegaProduct[pixel * 9 + 4] = c * dx * dy * domega;
	domegaProduct[pixel * 9 + 5] = c * dy * dz * domega;
	domegaProduct[pixel * 9 + 7] = c * dx * dz * domega;
	c = 0.315392;
	domegaProduct[pixel * 9 + 6] = c * (3 * dz * dz - 1) * domega;
	c = 0.546274;
	domegaProduct[pixel * 9 + 8] = c * (dx * dx - dy * dy) * domega;

}

void GPUComputeDomegaProduct2(float *cartesianCoord, float* sphericalCoord, float *domegaProduct, int width, int height)
{

	int blockSize, numBlocks, imageSize;
	imageSize = width * height;
	if(imageSize > 512)
		blockSize = 512;
	else
		blockSize = imageSize % 512;
	numBlocks = (imageSize / 512) + 1;

	computeDomegaProduct<<<numBlocks, blockSize>>>(cartesianCoord, sphericalCoord, domegaProduct, width, height);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
		printf("HDRImage.cu - GPUComputeDomegaProduct2: %s\n", hipGetErrorString(error));
	
	hipDeviceSynchronize();
}


__global__ void computeSHCoeffs(float *image, float *domegaProduct, float *SHCoeffs, int width, int height, int i) 
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > width * height) return;

	int pixel = idx;
	float u, v, r;
	float midWidth = width/2;
	float midHeight = height/2;
	int x = pixel % width;
	int y = pixel / width;

	u = (x - midWidth)/midWidth;
	v = (y - midHeight)/midHeight;
	r = sqrtf(u * u + v * v);
	if(r > 1.0) return;

	/*
	for(int ch = 0; ch < 3; ch++) {
		float imageValue = image[pixel * 3 + ch];
		atomicAdd(&SHCoeffs[0 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 0]);
		atomicAdd(&SHCoeffs[1 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 1]);
		atomicAdd(&SHCoeffs[2 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 2]);
		atomicAdd(&SHCoeffs[3 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 3]);
		atomicAdd(&SHCoeffs[4 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 4]);
		atomicAdd(&SHCoeffs[5 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 5]);
		atomicAdd(&SHCoeffs[7 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 7]);
		atomicAdd(&SHCoeffs[6 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 6]);
		atomicAdd(&SHCoeffs[8 * 3 + ch], imageValue * domegaProduct[pixel * 9 + 8]);
	}
	*/

	int sh = i / 3;
	int ch = i % 3;
	SHCoeffs[pixel] = image[pixel * 3 + ch] * domegaProduct[pixel * 9 + sh];
	
}

__global__ void scaleSHCoeffs(float *SHCoeffs, float scale) 
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	SHCoeffs[idx] *= scale;

}

__global__ void setZeroSHCoeffs(float *SHCoeffs) 
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	SHCoeffs[idx] = 0;

}

void GPUComputeSHCoeffs2(float *image, float *domegaProduct, float* SHCoeffs, float *auxSHCoeffs, float scale, int width, int height)
{

	int blockSize, numBlocks, imageSize;
	imageSize = width * height;
	if(imageSize > 512)
		blockSize = 512;
	else
		blockSize = imageSize % 512;
	numBlocks = (imageSize / 512) + 1;

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t stat;
	
	setZeroSHCoeffs<<<3, 9>>>(SHCoeffs);
	
	float sum[27];
	for(int i = 0; i < 27; i++) {
		setZeroSHCoeffs<<<numBlocks, blockSize>>>(auxSHCoeffs);
		computeSHCoeffs<<<numBlocks, blockSize>>>(image, domegaProduct, auxSHCoeffs, width, height, i);
		stat = hipblasSasum(handle, width * height, auxSHCoeffs, 1, &sum[i]);
	}
	
	hipblasDestroy(handle);
	hipMemcpy(SHCoeffs, sum, 27 * sizeof(float), hipMemcpyHostToDevice);
	scaleSHCoeffs<<<3, 9>>>(SHCoeffs, scale);
	
	/*
	computeSHCoeffs<<<numBlocks, blockSize>>>(image, domegaProduct, SHCoeffs, width, height, 0);
	scaleSHCoeffs<<<3, 9>>>(SHCoeffs, scale);
	*/
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
		printf("HDRImage.cu - GPUComputeSHCoeffs2: %s\n", hipGetErrorString(error));
	
	hipDeviceSynchronize();

}

__global__ void computeSphericalMap(float *cartesianCoord, float *SHCoeffs, float *image, int width, int height)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > width * height) return;

	int pixel = idx;
	
	float c[5];
	c[0] = 0.429043;
	c[1] = 0.511664;
	c[2] = 0.743125;
	c[3] = 0.886227;
	c[4] = 0.247708;

	float dx, dy, dz;
		
	int x = pixel % width;
	int y = pixel / width;

	dx = cartesianCoord[pixel * 3 + 0];
	dy = cartesianCoord[pixel * 3 + 1];
	dz = cartesianCoord[pixel * 3 + 2];
	
	if(dx == 0 && dy == 0 && dz == 0) {

		for(int ch = 0; ch < 3; ch++)
			image[pixel * 3 + ch] = 0;
		
	} else {

		for(int ch = 0; ch < 3; ch++) {

			image[pixel * 3 + ch] = c[0] * SHCoeffs[8 * 3 + ch] * (dx * dx - dy * dy) + 
				c[2] * SHCoeffs[6 * 3 + ch] * dz * dz +
				c[3] * SHCoeffs[0 * 3 + ch] -
				c[4] * SHCoeffs[6 * 3 + ch] +
				2 * c[0] * (SHCoeffs[4 * 3 + ch] * dx * dy + SHCoeffs[7 * 3 + ch] * dx * dz + SHCoeffs[5 * 3 + ch] * dy * dz) +
				2 * c[1] * (SHCoeffs[3 * 3 + ch] * dx + SHCoeffs[1 * 3 + ch] * dy + SHCoeffs[2 * 3 + ch] * dz);
			if(image[pixel * 3 + ch] < 0) image[pixel * 3 + ch] = 0;

		}

	}

}

void GPUComputeSphericalMap2(float *cartesianCoord, float *SHCoeffs, float *image, int width, int height)
{

	int blockSize, numBlocks, imageSize;
	imageSize = width * height;
	if(imageSize > 512)
		blockSize = 512;
	else
		blockSize = imageSize % 512;
	numBlocks = (imageSize / 512) + 1;

	computeSphericalMap<<<numBlocks, blockSize>>>(cartesianCoord, SHCoeffs, image, width, height);
	
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
		printf("HDRImage.cu - GPUComputeSHCoeffs2: %s\n", hipGetErrorString(error));
	
	hipDeviceSynchronize();

}
